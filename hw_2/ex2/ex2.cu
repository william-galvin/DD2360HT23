

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

#define DataType double

// Compute C = A * B
__global__ void gemm(DataType *A, DataType *B, DataType *C, int numARows,
                      int numAColumns, int numBRows, int numBColumns){
  //@@ Insert code to implement matrix multiplication here

}

int main(int argc, char **argv) {
  
  DataType *hostA; // The A matrix
  DataType *hostB; // The B matrix
  DataType *hostC; // The output C matrix
  DataType *resultRef; // The reference result
  DataType *deviceA;
  DataType *deviceB;
  DataType *deviceC;
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;
  int numCColumns;

  //@@ Insert code below to read in numARows, numAColumns, numBColumns from args

  printf("Input matrix dim (%d x %d) (%d x %d) (%d x %d)\n", numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);
  
  //@@ Insert code below to allocate Host memory for input and output

  
  //@@ Insert code below to initialize hostA and hostB to random numbers, and create reference result in CPU


  //@@ Insert code below to allocate GPU memory here


  //@@ Insert code to below to Copy memory to the GPU here


  //@@ Initialize the grid and block dimensions here


  //@@ Launch the GPU Kernel here


  //@@ Copy the GPU memory back to the CPU here


  //@@ Insert code below to compare the output with the reference


  //@@ Free the GPU memory here


  //@@ Free the CPU memory here


  return 0;
}

