#include "Particles.h"
#include "Alloc.h"
#include "CudaUtils.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

particles* device_alloc_particles(particles* h_particles) 
{
    particles* d_particles = (particles*)malloc(sizeof(particles));
    memcpy(d_particles, h_particles, sizeof(particles));
    particles* ret_val;

    long npmax = d_particles->npmax;

    hipMalloc(&d_particles->x, npmax * sizeof(FPpart));
    hipMalloc(&d_particles->y, npmax * sizeof(FPpart));
    hipMalloc(&d_particles->z, npmax * sizeof(FPpart));
    hipMalloc(&d_particles->u, npmax * sizeof(FPpart));
    hipMalloc(&d_particles->v, npmax * sizeof(FPpart));
    hipMalloc(&d_particles->w, npmax * sizeof(FPpart));
    hipMalloc(&ret_val, sizeof(particles));

    hipMemcpy(d_particles->x, h_particles->x, npmax * sizeof(FPpart), hipMemcpyHostToDevice);
    hipMemcpy(d_particles->y, h_particles->y, npmax * sizeof(FPpart), hipMemcpyHostToDevice);
    hipMemcpy(d_particles->z, h_particles->z, npmax * sizeof(FPpart), hipMemcpyHostToDevice);
    hipMemcpy(d_particles->u, h_particles->u, npmax * sizeof(FPpart), hipMemcpyHostToDevice);
    hipMemcpy(d_particles->v, h_particles->v, npmax * sizeof(FPpart), hipMemcpyHostToDevice);
    hipMemcpy(d_particles->w, h_particles->w, npmax * sizeof(FPpart), hipMemcpyHostToDevice);
    hipMemcpy(ret_val, d_particles, sizeof(particles), hipMemcpyHostToDevice);

    free(d_particles);
    return ret_val;
}

void device_free_particles(particles* d_part)
{
    particles* h_part = (particles*)malloc(sizeof(particles));
    hipMemcpy(h_part, d_part, sizeof(particles), hipMemcpyDeviceToHost);

    hipFree(h_part->x);
    hipFree(h_part->y);
    hipFree(h_part->z);
    hipFree(h_part->u);
    hipFree(h_part->v);
    hipFree(h_part->w);

    hipFree(d_part);
    free(h_part);
}

void copy_to_host_particles(particles* h_particles, particles* d_particles_)
{
    particles* d_particles = (particles*)malloc(sizeof(particles));
    hipMemcpy(d_particles, d_particles_, sizeof(particles), hipMemcpyDeviceToHost);

    long npmax = h_particles->npmax;
    hipMemcpy(h_particles->x, d_particles->x, npmax * sizeof(FPpart), hipMemcpyDeviceToHost);
    hipMemcpy(h_particles->y, d_particles->y, npmax * sizeof(FPpart), hipMemcpyDeviceToHost);
    hipMemcpy(h_particles->z, d_particles->z, npmax * sizeof(FPpart), hipMemcpyDeviceToHost);
    hipMemcpy(h_particles->u, d_particles->u, npmax * sizeof(FPpart), hipMemcpyDeviceToHost);
    hipMemcpy(h_particles->v, d_particles->v, npmax * sizeof(FPpart), hipMemcpyDeviceToHost);
    hipMemcpy(h_particles->w, d_particles->w, npmax * sizeof(FPpart), hipMemcpyDeviceToHost);

    free(d_particles);
}

EMfield* device_alloc_EMfield(EMfield* h_field, long len)
{
    EMfield* d_field = (EMfield*)malloc(sizeof(EMfield));
    memcpy(d_field, h_field, sizeof(EMfield));
    EMfield* ret_val;

    hipMalloc(&d_field->Ex_flat, len * sizeof(FPfield));
    hipMalloc(&d_field->Ey_flat, len * sizeof(FPfield));
    hipMalloc(&d_field->Ez_flat, len * sizeof(FPfield));
    hipMalloc(&d_field->Bxn_flat, len * sizeof(FPfield));
    hipMalloc(&d_field->Byn_flat, len * sizeof(FPfield));
    hipMalloc(&d_field->Bzn_flat, len * sizeof(FPfield));
    hipMalloc(&ret_val, sizeof(EMfield));

    hipMemcpy(d_field->Ex_flat, h_field->Ex_flat, len * sizeof(FPfield), hipMemcpyHostToDevice);
    hipMemcpy(d_field->Ey_flat, h_field->Ey_flat, len * sizeof(FPfield), hipMemcpyHostToDevice);
    hipMemcpy(d_field->Ez_flat, h_field->Ez_flat, len * sizeof(FPfield), hipMemcpyHostToDevice);
    hipMemcpy(d_field->Bxn_flat, h_field->Bxn_flat, len * sizeof(FPfield), hipMemcpyHostToDevice);
    hipMemcpy(d_field->Byn_flat, h_field->Byn_flat, len * sizeof(FPfield), hipMemcpyHostToDevice);
    hipMemcpy(d_field->Bzn_flat, h_field->Bzn_flat, len * sizeof(FPfield), hipMemcpyHostToDevice);
    hipMemcpy(ret_val, d_field, sizeof(EMfield), hipMemcpyHostToDevice);

    free(d_field);
    return ret_val;
}

void device_free_EMfield(EMfield* d_field) 
{
    EMfield* h_field = (EMfield*)malloc(sizeof(EMfield));
    hipMemcpy(h_field, d_field, sizeof(EMfield), hipMemcpyDeviceToHost);

    hipFree(h_field->Ex_flat);
    hipFree(h_field->Ey_flat);
    hipFree(h_field->Ez_flat);
    hipFree(h_field->Bxn_flat);
    hipFree(h_field->Byn_flat);
    hipFree(h_field->Bzn_flat);

    hipFree(d_field);
    free(h_field);
}

void copy_to_host_EMfield(EMfield* h_field, EMfield* d_field_, long len)
{
    EMfield* d_field = (EMfield*)malloc(sizeof(EMfield));
    hipMemcpy(d_field, d_field_, sizeof(EMfield), hipMemcpyDeviceToHost);

    hipMemcpy(h_field->Ex_flat, d_field->Ex_flat, len * sizeof(FPfield), hipMemcpyDeviceToHost);
    hipMemcpy(h_field->Ey_flat, d_field->Ey_flat, len * sizeof(FPfield), hipMemcpyDeviceToHost);
    hipMemcpy(h_field->Ez_flat, d_field->Ez_flat, len * sizeof(FPfield), hipMemcpyDeviceToHost);
    hipMemcpy(h_field->Bxn_flat, d_field->Bxn_flat, len * sizeof(FPfield), hipMemcpyDeviceToHost);
    hipMemcpy(h_field->Byn_flat, d_field->Byn_flat, len * sizeof(FPfield), hipMemcpyDeviceToHost);
    hipMemcpy(h_field->Bzn_flat, d_field->Bzn_flat, len * sizeof(FPfield), hipMemcpyDeviceToHost);

    free(d_field);
}   

grid* device_alloc_grid(grid* h_grid, long len)
{
    grid* d_grid = (grid*)malloc(sizeof(grid));
    memcpy(d_grid, h_grid, sizeof(grid));
    grid* ret_val;

    hipMalloc(&d_grid->XN_flat, len * sizeof(FPfield));
    hipMalloc(&d_grid->YN_flat, len * sizeof(FPfield));
    hipMalloc(&d_grid->ZN_flat, len * sizeof(FPfield));
    hipMalloc(&ret_val, sizeof(grid));

    hipMemcpy(d_grid->XN_flat, h_grid->XN_flat, len * sizeof(FPfield), hipMemcpyHostToDevice);
    hipMemcpy(d_grid->YN_flat, h_grid->YN_flat, len * sizeof(FPfield), hipMemcpyHostToDevice);
    hipMemcpy(d_grid->ZN_flat, h_grid->ZN_flat, len * sizeof(FPfield), hipMemcpyHostToDevice);
    hipMemcpy(ret_val, d_grid, sizeof(grid), hipMemcpyHostToDevice);
    
    free(d_grid);
    return ret_val;
}

void copy_to_host_grid(grid* h_grid, grid* d_grid_, long len) 
{
    grid* d_grid = (grid*)malloc(sizeof(grid));
    hipMemcpy(d_grid, d_grid_, sizeof(grid), hipMemcpyDeviceToHost);

    hipMemcpy(h_grid->XN_flat, d_grid->XN_flat, len * sizeof(FPfield), hipMemcpyDeviceToHost);
    hipMemcpy(h_grid->YN_flat, d_grid->YN_flat, len * sizeof(FPfield), hipMemcpyDeviceToHost);
    hipMemcpy(h_grid->ZN_flat, d_grid->ZN_flat, len * sizeof(FPfield), hipMemcpyDeviceToHost);

    free(d_grid);
}

void device_free_grid(grid* d_grid)
{
    grid* h_grid = (grid*)malloc(sizeof(grid));
    hipMemcpy(h_grid, d_grid, sizeof(grid), hipMemcpyDeviceToHost);

    hipFree(h_grid->XN_flat);
    hipFree(h_grid->YN_flat);
    hipFree(h_grid->ZN_flat);

    hipFree(d_grid);
    free(h_grid);
}

parameters* device_alloc_parameters(parameters* h_param)
{
    parameters* d_param;
    hipMalloc(&d_param, sizeof(parameters));
    hipMemcpy(d_param, h_param, sizeof(parameters), hipMemcpyHostToDevice);
    return d_param;
}

void copy_to_host_parameters(parameters* h_param, parameters* d_param)
{
    hipMemcpy(h_param, d_param, sizeof(parameters), hipMemcpyDeviceToHost);
}

void device_free_parameters(parameters* d_param)
{
    hipFree(d_param);
}
